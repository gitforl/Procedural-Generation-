#include "hip/hip_runtime.h"
#include "../utilities/descriptorDistance.cuh"
#include "utility.cuh"

namespace SIUTILITIES {

    __device__ float computeSpinImagePairCorrelationGPU(
            ShapeDescriptor::SpinImageDescriptor* descriptors,
            ShapeDescriptor::SpinImageDescriptor* otherDescriptors,
            size_t spinImageIndex,
            size_t otherImageIndex,
            float averageX, float averageY) {

        float threadSquaredSumX = 0;
        float threadSquaredSumY = 0;
        float threadMultiplicativeSum = 0;

        spinImagePixelType pixelValueX;
        spinImagePixelType pixelValueY;

        for (int y = 0; y < spinImageWidthPixels; y++)
        {
            const int warpSize = 32;
            for (int x = threadIdx.x % 32; x < spinImageWidthPixels; x += warpSize)
            {
                pixelValueX = descriptors[spinImageIndex].contents[y * spinImageWidthPixels + x];
                pixelValueY = otherDescriptors[otherImageIndex].contents[y * spinImageWidthPixels + x];

                float deltaX = float(pixelValueX) - averageX;
                float deltaY = float(pixelValueY) - averageY;

                threadSquaredSumX += deltaX * deltaX;
                threadSquaredSumY += deltaY * deltaY;
                threadMultiplicativeSum += deltaX * deltaY;
            }
        }

        float squaredSumX = float(sqrt(warpAllReduceSum(threadSquaredSumX)));
        float squaredSumY = float(sqrt(warpAllReduceSum(threadSquaredSumY)));
        float multiplicativeSum = warpAllReduceSum(threadMultiplicativeSum);

        float correlation = multiplicativeSum / (squaredSumX * squaredSumY);

        return correlation;
    }
}

__global__
void ComputePairDistance(
    ShapeDescriptor::gpu::array<ShapeDescriptor::SpinImageDescriptor> descriptors, 
    ShapeDescriptor::gpu::array<ShapeDescriptor::SpinImageDescriptor> otherDescriptors,
    int *results
)
{
    const size_t descriptorIndex = blockIdx.x;

    static_assert(spinImageWidthPixels % 32 == 0, "This kernel assumes the image is a multiple of the warp size wide");


    float threadSquaredSum = 0;

    for(unsigned int i = threadIdx.x; i < spinImageWidthPixels * spinImageWidthPixels; i += blockDim.x) {
        spinImagePixelType descriptorPixelValue = descriptors[descriptorIndex].contents[i];
        spinImagePixelType correspondingPixelValue = otherDescriptors[descriptorIndex].contents[i];
        spinImagePixelType pixelDelta = descriptorPixelValue - correspondingPixelValue;
        threadSquaredSum += pixelDelta * pixelDelta;
    }

    float totalSquaredSum = warpAllReduceSum(threadSquaredSum);

    if(threadIdx.x == 0) {
        results[descriptorIndex] = totalSquaredSum;
    }
}

void DescriptorDistance::SI::ComputePairWise(
    ShapeDescriptor::gpu::array<ShapeDescriptor::SpinImageDescriptor> needleDescriptors,
    ShapeDescriptor::gpu::array<ShapeDescriptor::SpinImageDescriptor> haystackDescriptors
)
{

    const unsigned int numDescriptors = needleDescriptors.length;

    int *results;
    hipMallocManaged(&results, numDescriptors * sizeof(int));

    ComputePairDistance<<<numDescriptors, 32 >>>(needleDescriptors, haystackDescriptors, results);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    for(unsigned int i = 0; i < numDescriptors; i++)
    {
        std::cout << "Distance at " << i << ": " << results[i] << std::endl;
    }

        unsigned int exponent = 0;

    for (unsigned int i = 0; i < numDescriptors; i++)
    {
        while (pow(2, exponent) < results[i])
            exponent++;
        // std::cout << "Distance at " << i << ": " << results[i] << std::endl;
    }

    float min = 0.0f, max = pow(2,exponent);//1024.0f;

    std::vector<float> fResults;
    fResults.reserve(numDescriptors);
    for (size_t i = 0; i < numDescriptors; i++)
        fResults.emplace_back(results[i]);

    showHistogram(fResults.data(), numDescriptors, 16, max, min, "../images/json/si/pairwise_combined.json");

    hipFree(results);

}


__global__
void ComputeCrossDistance(
    ShapeDescriptor::gpu::array<ShapeDescriptor::SpinImageDescriptor> descriptors, 
    ShapeDescriptor::gpu::array<ShapeDescriptor::SpinImageDescriptor> otherDescriptors,
    DescriptorDistance::intDistances *results
)
{
 
    const size_t blockYZIndex = blockIdx.y + blockIdx.z * gridDim.z;
    
    const size_t descriptorIndex = blockIdx.x;
    const size_t otherDescriptorIndex = blockYZIndex < otherDescriptors.length ? blockYZIndex : otherDescriptors.length - 1;
    const int laneIndex = threadIdx.x;

    static_assert(spinImageWidthPixels % 32 == 0, "This kernel assumes the image is a multiple of the warp size wide");


    float threadSquaredSum = 0;

    for(unsigned int i = threadIdx.x; i < spinImageWidthPixels * spinImageWidthPixels; i += blockDim.x) {
        spinImagePixelType descriptorPixelValue = descriptors[descriptorIndex].contents[i];
        spinImagePixelType correspondingPixelValue = otherDescriptors[otherDescriptorIndex].contents[i];
        spinImagePixelType pixelDelta = descriptorPixelValue - correspondingPixelValue;
        threadSquaredSum += pixelDelta * pixelDelta;
    }

    float totalSquaredSum = warpAllReduceSum(threadSquaredSum);

    if(laneIndex == 0 && blockYZIndex < otherDescriptors.length)
    {
        atomicMin(&results[descriptorIndex].min, totalSquaredSum);
        atomicMax(&results[descriptorIndex].max, totalSquaredSum);
        atomicAdd(&results[descriptorIndex].avg, totalSquaredSum);
    }
}

void DescriptorDistance::SI::ComputeCrossWise(
    ShapeDescriptor::gpu::array<ShapeDescriptor::SpinImageDescriptor> needleDescriptors,
    ShapeDescriptor::gpu::array<ShapeDescriptor::SpinImageDescriptor> haystackDescriptors
    )
{
    const unsigned int numDescriptors = needleDescriptors.length;
    const unsigned int numOtherDescriptors = haystackDescriptors.length;

    intDistances *results;
    hipMallocManaged(&results, numDescriptors * sizeof(intDistances));

    fillDistances(results, numDescriptors);

    const auto [numY, numZ] = findCorrectDimensions(numOtherDescriptors);

    ComputeCrossDistance<<<dim3(numDescriptors, numY, numZ), 32 >>>(needleDescriptors, haystackDescriptors, results);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    unsigned int exponent = 0;

    for(unsigned int i = 0; i < numDescriptors; i++)
    {
        while(pow(2,exponent) < results[i].min || pow(2,exponent) < (results[i].avg / numOtherDescriptors) || pow(2,exponent) < results[i].max)
            exponent++;
        // std::cout << "Distance at " << i << ": " << results[i] << std::endl;
    }

    float min = 0.0f, max = pow(2,exponent);//1024.0f;

    std::vector<float> fResults;
    fResults.reserve(numDescriptors);
    for (size_t i = 0; i < numDescriptors; i++)
        fResults.emplace_back(results[i].min);

    showHistogram(fResults.data(), numDescriptors, 16, max, min, "../images/json/si/I_min.json");

    fResults.clear();
    for (size_t i = 0; i < numDescriptors; i++)
        fResults.emplace_back((results[i].avg / numOtherDescriptors));

    showHistogram(fResults.data(), numDescriptors, 16, max, min, "../images/json/si/I_avg.json");

    fResults.clear();
    for (size_t i = 0; i < numDescriptors; i++)
        fResults.emplace_back(results[i].max);

    showHistogram(fResults.data(), numDescriptors, 16, max, min, "../images/json/si/I_max.json");
    
    hipFree(results);
}

__global__
void ComputeCrossDistanceWithThreshold(
    ShapeDescriptor::gpu::array<ShapeDescriptor::SpinImageDescriptor> descriptors, 
    ShapeDescriptor::gpu::array<ShapeDescriptor::SpinImageDescriptor> otherDescriptors,
    ShapeDescriptor::gpu::array<float> thresholds,
    unsigned int *results
)
{
    const size_t blockYZIndex = blockIdx.y + blockIdx.z * gridDim.z;
    
    const size_t descriptorIndex = blockIdx.x;
    const size_t otherDescriptorIndex = blockYZIndex < otherDescriptors.length ? blockYZIndex : otherDescriptors.length - 1;
    const int laneIndex = threadIdx.x;

    static_assert(spinImageWidthPixels % 32 == 0, "This kernel assumes the image is a multiple of the warp size wide");


    float threadSquaredSum = 0;

    for(unsigned int i = threadIdx.x; i < spinImageWidthPixels * spinImageWidthPixels; i += blockDim.x) {
        spinImagePixelType descriptorPixelValue = descriptors[descriptorIndex].contents[i];
        spinImagePixelType correspondingPixelValue = otherDescriptors[otherDescriptorIndex].contents[i];
        spinImagePixelType pixelDelta = descriptorPixelValue - correspondingPixelValue;
        threadSquaredSum += pixelDelta * pixelDelta;
    }

    float totalSquaredSum = warpAllReduceSum(threadSquaredSum);

    if(laneIndex == 0 && totalSquaredSum < thresholds[descriptorIndex]  && blockYZIndex < otherDescriptors.length)
    {
        atomicAdd(&results[descriptorIndex], 1);
    }
}

void DescriptorDistance::SI::ComputeCrossWiseWithThreshold(
    ShapeDescriptor::gpu::array<ShapeDescriptor::SpinImageDescriptor> needleDescriptors,
    ShapeDescriptor::gpu::array<ShapeDescriptor::SpinImageDescriptor> haystackDescriptors,
    ShapeDescriptor::gpu::array<float> thresholds
)
{
    const unsigned int numDescriptors = needleDescriptors.length;
    const unsigned int numOtherDescriptors = haystackDescriptors.length;

    unsigned int *results;
    hipMallocManaged(&results, numDescriptors * sizeof(unsigned int));

    fillZeros(results, numDescriptors);

    const auto [numY, numZ] = findCorrectDimensions(numOtherDescriptors);

    ComputeCrossDistanceWithThreshold<<<dim3(numDescriptors, numY, numZ), 32 >>>(needleDescriptors, haystackDescriptors, thresholds, results);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    for(unsigned int i = 0; i < numDescriptors; i++)
    {
        std::cout << "Number of distances below threshold " << i << ": " << results[i] << std::endl;
    }

    
    hipFree(results);
}
   

