#include "hip/hip_runtime.h"
#include "../utilities/descriptorDistance.cuh"
#include "utility.cuh"

namespace RICIUTILITES
{

    const int indexBasedWarpCount = 16;

    __device__ int computeImageSquaredSumGPU(const ShapeDescriptor::RICIDescriptor &needleImage)
    {

        const int spinImageElementCount = spinImageWidthPixels * spinImageWidthPixels;
        const int laneIndex = threadIdx.x % 32;

        unsigned int threadSquaredSum = 0;

        static_assert(spinImageWidthPixels % 32 == 0, "This kernel assumes an image whose width is a multiple of the warp size");

        // Scores are computed one row at a time.
        // We differentiate between rows to ensure the final pixel of the previous row does not
        // affect the first pixel of the next one.
        for (int pixel = 0; pixel < spinImageElementCount; pixel++)
        {
            radialIntersectionCountImagePixelType previousWarpLastNeedlePixelValue = 0;
            radialIntersectionCountImagePixelType currentNeedlePixelValue =
                needleImage.contents[pixel];

            int targetThread;
            if (laneIndex > 0)
            {
                targetThread = laneIndex - 1;
            }
            else if (pixel > 0)
            {
                targetThread = 31;
            }
            else
            {
                targetThread = 0;
            }

            radialIntersectionCountImagePixelType threadNeedleValue = 0;

            if (laneIndex == 31)
            {
                threadNeedleValue = previousWarpLastNeedlePixelValue;
            }
            else
            {
                threadNeedleValue = currentNeedlePixelValue;
            }

            radialIntersectionCountImagePixelType previousNeedlePixelValue = __shfl_sync(0xFFFFFFFF, threadNeedleValue, targetThread);
            int needleDelta = int(currentNeedlePixelValue) - int(previousNeedlePixelValue);

            threadSquaredSum += unsigned(needleDelta * needleDelta);
        }

        int squaredSum = warpAllReduceSum(threadSquaredSum);

        return squaredSum;
    }

    __device__ size_t compareConstantRadialIntersectionCountImagePairGPU(
        const ShapeDescriptor::RICIDescriptor *needleImages,
        const size_t needleImageIndex,
        const ShapeDescriptor::RICIDescriptor *haystackImages,
        const size_t haystackImageIndex)
    {

        const int laneIndex = threadIdx.x % 32;

        // Assumption: there will never be an intersection count over 65535 (which would cause this to overflow)
        size_t threadDeltaSquaredSum = 0;

        static_assert(spinImageWidthPixels % 32 == 0, "This kernel assumes an image whose width is a multiple of the warp size");

        // Scores are computed one row at a time.
        // We differentiate between rows to ensure the final pixel of the previous row does not
        // affect the first pixel of the next one.
        for (int row = 0; row < spinImageWidthPixels; row++)
        {
            // Each thread processes one pixel, a warp processes therefore 32 pixels per iteration
            for (int pixel = laneIndex; pixel < spinImageWidthPixels; pixel += warpSize)
            {
                radialIntersectionCountImagePixelType currentNeedlePixelValue =
                    needleImages[needleImageIndex].contents[row * spinImageWidthPixels + pixel];
                radialIntersectionCountImagePixelType currentHaystackPixelValue =
                    haystackImages[haystackImageIndex].contents[row * spinImageWidthPixels + pixel];

                // This bit handles the case where an image is completely constant.
                // In that case, we use the absolute sum of squares as a distance function instead
                int imageDelta = int(currentNeedlePixelValue) - int(currentHaystackPixelValue);
                threadDeltaSquaredSum += unsigned(imageDelta * imageDelta); // TODO: size_t?
            }
        }

        // image is constant.
        // In those situations, imageScore would always be 0
        // So we use an unfiltered squared sum instead
        size_t imageScore = warpAllReduceSum(threadDeltaSquaredSum);

        return imageScore;
    }

    __device__ int compareRadialIntersectionCountImagePairGPU(
        const ShapeDescriptor::RICIDescriptor *needleImages,
        const size_t needleImageIndex,
        const ShapeDescriptor::RICIDescriptor *haystackImages,
        const size_t haystackImageIndex,
        const int distanceToBeat = INT_MAX)
    {

        int threadScore = 0;
        const int laneIndex = threadIdx.x % 32;

        static_assert(spinImageWidthPixels % 32 == 0, "This kernel assumes an image whose width is a multiple of the warp size");

        // Scores are computed one row at a time.
        // We differentiate between rows to ensure the final pixel of the previous row does not
        // affect the first pixel of the next one.
        for (int row = 0; row < spinImageWidthPixels; row++)
        {
            radialIntersectionCountImagePixelType previousWarpLastNeedlePixelValue = 0;
            radialIntersectionCountImagePixelType previousWarpLastHaystackPixelValue = 0;
            // Each thread processes one pixel, a warp processes therefore 32 pixels per iteration
            for (int pixel = laneIndex; pixel < spinImageWidthPixels; pixel += warpSize)
            {
                radialIntersectionCountImagePixelType currentNeedlePixelValue =
                    needleImages[needleImageIndex].contents[row * spinImageWidthPixels + pixel];
                radialIntersectionCountImagePixelType currentHaystackPixelValue =
                    haystackImages[haystackImageIndex].contents[row * spinImageWidthPixels + pixel];

                // To save on memory bandwidth, we use shuffle instructions to pass around other values needed by the
                // distance computation. We first need to use some logic to determine which thread should read from which
                // other thread.
                int targetThread;
                if (laneIndex > 0)
                {
                    // Each thread reads from the previous one
                    targetThread = laneIndex - 1;
                }
                // For these last two: lane index is 0. The first pixel of each row receives special treatment, as
                // there is no pixel left of it you can compute a difference over
                else if (pixel > 0)
                {
                    // If pixel is not the first pixel in the row, we read the difference value from the previous iteration
                    targetThread = 31;
                }
                else
                {
                    // If the pixel is the leftmost pixel in the row, we give targetThread a dummy value that will always
                    // result in a pixel delta of zero: our own thread with ID 0.
                    targetThread = 0;
                }

                radialIntersectionCountImagePixelType threadNeedleValue = 0;
                radialIntersectionCountImagePixelType threadHaystackValue = 0;

                // Here we determine the outgoing value of the shuffle.
                // If we're the last thread in the warp, thread 0 will read from us if we're not processing the first batch
                // of 32 pixels in the row. Since in that case thread 0 will read from itself, we can simplify that check
                // into whether we are lane 31.
                if (laneIndex == 31)
                {
                    threadNeedleValue = previousWarpLastNeedlePixelValue;
                    threadHaystackValue = previousWarpLastHaystackPixelValue;
                }
                else
                {
                    threadNeedleValue = currentNeedlePixelValue;
                    threadHaystackValue = currentHaystackPixelValue;
                }

                // Exchange "previous pixel" values through shuffle instructions
                radialIntersectionCountImagePixelType previousNeedlePixelValue = __shfl_sync(0xFFFFFFFF, threadNeedleValue, targetThread);
                radialIntersectionCountImagePixelType previousHaystackPixelValue = __shfl_sync(0xFFFFFFFF, threadHaystackValue,
                                                                                               targetThread);

                // The distance measure this function computes is based on deltas between pairs of pixels
                int needleDelta = int(currentNeedlePixelValue) - int(previousNeedlePixelValue);
                int haystackDelta = int(currentHaystackPixelValue) - int(previousHaystackPixelValue);

                // This if statement makes a massive difference in the clutter resistant performance of this method
                // It only counts least squares differences if the needle image has a change in intersection count
                // Which is usually something very specific to that object.
                if (needleDelta != 0)
                {
                    threadScore += (needleDelta - haystackDelta) * (needleDelta - haystackDelta);
                }

                // This only matters for thread 31, so no need to broadcast it using a shuffle instruction
                previousWarpLastNeedlePixelValue = currentNeedlePixelValue;
                previousWarpLastHaystackPixelValue = currentHaystackPixelValue;
            }
#if ENABLE_RICI_COMPARISON_EARLY_EXIT
            // At the end of each block of 8 rows, check whether we can do an early exit
            // This also works for the constant image
            if (row != (spinImageWidthPixels - 1))
            {
                int intermediateDistance = warpAllReduceSum(threadScore);
                if (intermediateDistance >= distanceToBeat)
                {
                    return intermediateDistance;
                }
            }
#endif
        }

        int imageScore = warpAllReduceSum(threadScore);

        return imageScore;
    }

}

__global__ void ComputePairDistance(
    ShapeDescriptor::gpu::array<ShapeDescriptor::RICIDescriptor> descriptors,
    ShapeDescriptor::gpu::array<ShapeDescriptor::RICIDescriptor> otherDescriptors,
    int *results)
{
    const size_t descriptorIndex = blockIdx.x;

    static_assert(spinImageWidthPixels % 32 == 0, "This kernel assumes the image is a multiple of the warp size wide");

    int distanceScore;
    int needleSquaredSum = RICIUTILITES::computeImageSquaredSumGPU(descriptors[descriptorIndex]);
    bool needleImageIsConstant = needleSquaredSum == 0;

    if (!needleImageIsConstant)
    {
        distanceScore = RICIUTILITES::compareRadialIntersectionCountImagePairGPU(
            descriptors.content, descriptorIndex,
            otherDescriptors.content, descriptorIndex);
    }
    else
    {
        distanceScore = RICIUTILITES::compareConstantRadialIntersectionCountImagePairGPU(
            descriptors.content, descriptorIndex,
            otherDescriptors.content, descriptorIndex);
    }

    if (threadIdx.x == 0)
    {
        results[descriptorIndex] = distanceScore;
    }
}

void DescriptorDistance::RICI::ComputePairWise(
    ShapeDescriptor::gpu::array<ShapeDescriptor::RICIDescriptor> needleDescriptors,
    ShapeDescriptor::gpu::array<ShapeDescriptor::RICIDescriptor> haystackDescriptors)
{

    const unsigned int numDescriptors = needleDescriptors.length;

    int *results;
    hipMallocManaged(&results, numDescriptors * sizeof(int));

    ComputePairDistance<<<numDescriptors, 32>>>(needleDescriptors, haystackDescriptors, results);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    unsigned int exponent = 0;

    for (unsigned int i = 0; i < numDescriptors; i++)
    {
        while (pow(2, exponent) < results[i])
            exponent++;
        // std::cout << "Distance at " << i << ": " << results[i] << std::endl;
    }

    float min = 0.0f, max = 2048; // pow(2,exponent);//1024.0f;

    std::vector<float> fResults;
    fResults.reserve(numDescriptors);
    for (size_t i = 0; i < numDescriptors; i++)
        fResults.emplace_back(results[i]);

    showHistogram(fResults.data(), numDescriptors, 16, max, min, "../images/json/rici/combined_pairwise.json");

    hipFree(results);
}

__global__ void ComputeCrossDistance(
    ShapeDescriptor::gpu::array<ShapeDescriptor::RICIDescriptor> descriptors,
    ShapeDescriptor::gpu::array<ShapeDescriptor::RICIDescriptor> otherDescriptors,
    DescriptorDistance::intDistances *results)
{

    const size_t blockYZIndex = blockIdx.y + blockIdx.z * gridDim.z;

    const size_t descriptorIndex = blockIdx.x;
    const size_t otherDescriptorIndex = blockYZIndex < otherDescriptors.length ? blockYZIndex : otherDescriptors.length - 1;
    const int laneIndex = threadIdx.x;

    static_assert(spinImageWidthPixels % 32 == 0, "This kernel assumes the image is a multiple of the warp size wide");

    int distanceScore;
    int needleSquaredSum = RICIUTILITES::computeImageSquaredSumGPU(descriptors[descriptorIndex]);
    bool needleImageIsConstant = needleSquaredSum == 0;

    if (!needleImageIsConstant)
    {
        distanceScore = RICIUTILITES::compareRadialIntersectionCountImagePairGPU(
            descriptors.content, descriptorIndex,
            otherDescriptors.content, otherDescriptorIndex);
    }
    else
    {
        distanceScore = RICIUTILITES::compareConstantRadialIntersectionCountImagePairGPU(
            descriptors.content, descriptorIndex,
            otherDescriptors.content, otherDescriptorIndex);
    }

    if (laneIndex == 0 && blockYZIndex < otherDescriptors.length)
    {
        atomicMin(&results[descriptorIndex].min, distanceScore);
        atomicMax(&results[descriptorIndex].max, distanceScore);
        atomicAdd(&results[descriptorIndex].avg, distanceScore);
    }
}

void DescriptorDistance::RICI::ComputeCrossWise(
    ShapeDescriptor::gpu::array<ShapeDescriptor::RICIDescriptor> needleDescriptors,
    ShapeDescriptor::gpu::array<ShapeDescriptor::RICIDescriptor> haystackDescriptors)
{
    const unsigned int numDescriptors = needleDescriptors.length;
    const unsigned int numOtherDescriptors = haystackDescriptors.length;

    intDistances *results;
    hipMallocManaged(&results, numDescriptors * sizeof(intDistances));

    fillDistances(results, numDescriptors);

    const auto [numY, numZ] = findCorrectDimensions(numOtherDescriptors);

    ComputeCrossDistance<<<dim3(numDescriptors, numY, numZ), 32>>>(needleDescriptors, haystackDescriptors, results);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    unsigned int numZero = 0;

    for (unsigned int i = 0; i < numDescriptors; i++)
    {
        // std::cout << "distances at " << i << ", min:" << results[i].min << ", max:" << results[i].max << ", avg:" << (results[i].avg / numOtherDescriptors) << std::endl;
        // if(results[i].min > (results[i].avg / numOtherDescriptors))
        //     std::cout << "avg distance at " << i << "below min" << std::endl;
        // if(results[i].max < (results[i].avg / numOtherDescriptors))
        //     std::cout << "avg distance at " << i << "above max" << std::endl;
        // if(results[i].min == 0)
        //     numZero++;
    }

    // std::cout << "Number of zero distances: " << numZero << std::endl;

    unsigned int exponent = 0;

    for(unsigned int i = 0; i < numDescriptors; i++)
    {
        while(pow(2,exponent) < results[i].min || pow(2,exponent) < (results[i].avg / numOtherDescriptors) || pow(2,exponent) < results[i].max)
            exponent++;
        // std::cout << "Distance at " << i << ": " << results[i] << std::endl;
    }

    float min = 0.0f, max = pow(2,exponent);//1024.0f;

    std::vector<float> fResults;
    fResults.reserve(numDescriptors);
    for (size_t i = 0; i < numDescriptors; i++)
        fResults.emplace_back(results[i].min);

    showHistogram(fResults.data(), numDescriptors, 16, max, min, "../images/json/combined/histogram_E_min_RICI.json");

    fResults.clear();
    for (size_t i = 0; i < numDescriptors; i++)
        fResults.emplace_back((results[i].avg / numOtherDescriptors));

    showHistogram(fResults.data(), numDescriptors, 16, max, min, "../images/json/combined/histogram_E_avg_RICI.json");

    fResults.clear();
    for (size_t i = 0; i < numDescriptors; i++)
        fResults.emplace_back(results[i].max);

    showHistogram(fResults.data(), numDescriptors, 16, max, min, "../images/json/combined/histogram_E_max_RICI.json");

    hipFree(results);
}

__global__ void ComputeCrossDistanceWithThreshold(
    ShapeDescriptor::gpu::array<ShapeDescriptor::RICIDescriptor> descriptors,
    ShapeDescriptor::gpu::array<ShapeDescriptor::RICIDescriptor> otherDescriptors,
    ShapeDescriptor::gpu::array<float> thresholds,
    unsigned int *results)
{
    const size_t blockYZIndex = blockIdx.y + blockIdx.z * gridDim.z;

    const size_t descriptorIndex = blockIdx.x;
    const size_t otherDescriptorIndex = blockYZIndex < otherDescriptors.length ? blockYZIndex : otherDescriptors.length - 1;
    const int laneIndex = threadIdx.x;

    static_assert(spinImageWidthPixels % 32 == 0, "This kernel assumes the image is a multiple of the warp size wide");

    int distanceScore;
    int needleSquaredSum = RICIUTILITES::computeImageSquaredSumGPU(descriptors[descriptorIndex]);
    bool needleImageIsConstant = needleSquaredSum == 0;

    if (!needleImageIsConstant)
    {
        distanceScore = RICIUTILITES::compareRadialIntersectionCountImagePairGPU(
            descriptors.content, descriptorIndex,
            otherDescriptors.content, otherDescriptorIndex);
    }
    else
    {
        distanceScore = RICIUTILITES::compareConstantRadialIntersectionCountImagePairGPU(
            descriptors.content, descriptorIndex,
            otherDescriptors.content, otherDescriptorIndex);
    }

    if (laneIndex == 0 && distanceScore < thresholds[descriptorIndex] && blockYZIndex < otherDescriptors.length)
    {
        atomicAdd(&results[descriptorIndex], 1);
    }
}

void DescriptorDistance::RICI::ComputeCrossWiseWithThreshold(
    ShapeDescriptor::gpu::array<ShapeDescriptor::RICIDescriptor> needleDescriptors,
    ShapeDescriptor::gpu::array<ShapeDescriptor::RICIDescriptor> haystackDescriptors,
    ShapeDescriptor::gpu::array<float> thresholds)
{
    const unsigned int numDescriptors = needleDescriptors.length;
    const unsigned int numOtherDescriptors = haystackDescriptors.length;

    unsigned int *results;
    hipMallocManaged(&results, numDescriptors * sizeof(unsigned int));

    fillZeros(results, numDescriptors);

    const auto [numY, numZ] = findCorrectDimensions(numOtherDescriptors);

    ComputeCrossDistanceWithThreshold<<<dim3(numDescriptors, numY, numZ), 32>>>(needleDescriptors, haystackDescriptors, thresholds, results);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    for (unsigned int i = 0; i < numDescriptors; i++)
    {
        std::cout << "Number of distances below threshold " << i << ": " << results[i] << std::endl;
    }

    hipFree(results);
}
