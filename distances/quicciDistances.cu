#include "hip/hip_runtime.h"
#include "../utilities/descriptorDistance.cuh"
#include "utility.cuh"

__inline__ __device__ unsigned int getChunkAt(const ShapeDescriptor::QUICCIDescriptor* image, const size_t imageIndex, const int chunkIndex) {
    return image[imageIndex].contents[chunkIndex];
}

__inline__ __device__ int computeImageSumGPU(
        const ShapeDescriptor::QUICCIDescriptor* needleImages,
        const size_t imageIndex) 
{

    const int laneIndex = threadIdx.x % 32;

    unsigned int threadSum = 0;

    static_assert(spinImageWidthPixels % 32 == 0, "This kernel assumes images are multiples of warp size wide");

    for (int chunk = laneIndex; chunk < uintsPerQUICCImage; chunk += warpSize) {
        unsigned int needleChunk = getChunkAt(needleImages, imageIndex, chunk);
        threadSum += __popc(needleChunk);
    }

    int sum = warpAllReduceSum(threadSum);

    return sum;
}

__device__ __forceinline__ float computeQUICCIThreadDistance(
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> descriptors, 
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> otherDescriptors,
    const size_t descriptorIndex,
    const size_t otherDescriptorsIndex
)
{
    const int laneIndex = threadIdx.x;

    int referenceImageBitCount = computeImageSumGPU(descriptors.content, descriptorIndex);
    ShapeDescriptor::utilities::HammingWeights hammingWeights = ShapeDescriptor::utilities::computeWeightedHammingWeights(referenceImageBitCount, spinImageWidthPixels * spinImageWidthPixels);

    bool needleImageIsConstant = referenceImageBitCount == 0;

    float threadWeightedHammingDistance = 0;

    auto chunk = laneIndex;

    if(!needleImageIsConstant) 
    {

        unsigned int needleChunk = getChunkAt(descriptors.content, descriptorIndex, chunk);
        unsigned int haystackChunk = getChunkAt(otherDescriptors.content, otherDescriptorsIndex, chunk);

        threadWeightedHammingDistance += ShapeDescriptor::utilities::computeChunkWeightedHammingDistance(hammingWeights, needleChunk, haystackChunk);

    } else 
    {
        unsigned int haystackChunk = getChunkAt(otherDescriptors.content, descriptorIndex, chunk);

        threadWeightedHammingDistance += float(__popc(haystackChunk));
        
    }

    return threadWeightedHammingDistance;
}

__global__
void ComputePairDistance(
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> descriptors, 
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> otherDescriptors,
    float *results
)
{
    const size_t descriptorIndex = blockIdx.x;
    const int laneIndex = threadIdx.x;


    float threadWeightedHammingDistance = computeQUICCIThreadDistance(descriptors, otherDescriptors, descriptorIndex, descriptorIndex);

    __syncthreads();

    float weightedHammingDistance = warpAllReduceSum(threadWeightedHammingDistance);

    if(laneIndex == 0)
    {
        results[blockIdx.x] = weightedHammingDistance;
    }
}

void DescriptorDistance::QUICCI::ComputePairWise(
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> needleDescriptors,
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> haystackDescriptors
)
{

    const unsigned int numDescriptors = needleDescriptors.length;

    float *results;
    hipMallocManaged(&results, numDescriptors * sizeof(float));

    ComputePairDistance<<<numDescriptors, 32 >>>(needleDescriptors, haystackDescriptors, results);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());


    for(unsigned int i = 0; i < numDescriptors; i++)
    {
        // std::cout << "Distance at" << i << ": " << results[i] << std::endl;
        // if(results[i] < min) min = results[i];
        // if(results[i] > max) max = results[i];
    }


    float min = 0.0f, max = 1024.0f;
    showHistogram(results, numDescriptors, 16, max, min, "../images/json/clutter/histogram_C_4.json");

    hipFree(results);

}

__global__
void ComputeCrossDistance(
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> descriptors, 
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> otherDescriptors,
    DescriptorDistance::distances *results
)
{
 
    const size_t blockYZIndex = blockIdx.y + blockIdx.z * gridDim.z;
    
    const size_t descriptorIndex = blockIdx.x;
    const size_t otherDescriptorIndex = blockYZIndex < otherDescriptors.length ? blockYZIndex : otherDescriptors.length - 1;
    const int laneIndex = threadIdx.x;


    float threadWeightedHammingDistance = computeQUICCIThreadDistance(descriptors, otherDescriptors, descriptorIndex, otherDescriptorIndex);

    __syncthreads();

    float weightedHammingDistance = warpAllReduceSum(threadWeightedHammingDistance);

    if(laneIndex == 0 && blockYZIndex < otherDescriptors.length)
    {
        atomicMinFloat(&results[descriptorIndex].min, weightedHammingDistance);
        atomicMaxFloat(&results[descriptorIndex].max, weightedHammingDistance);
        atomicAdd(&results[descriptorIndex].avg, weightedHammingDistance);
    }
}

void DescriptorDistance::QUICCI::ComputeCrossWise(
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> needleDescriptors,
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> haystackDescriptors
    )
{
    const unsigned int numDescriptors = needleDescriptors.length;
    const unsigned int numOtherDescriptors = haystackDescriptors.length;

    distances *results;
    hipMallocManaged(&results, numDescriptors * sizeof(distances));

    fillDistances(results, numDescriptors);

    const auto [numY, numZ] = findCorrectDimensions(numOtherDescriptors);

    ComputeCrossDistance<<<dim3(numDescriptors, numY, numZ), 32 >>>(needleDescriptors, haystackDescriptors, results);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    // unsigned int numZero = 0;

    distances overallDistances = {std::numeric_limits<float>::max(), 0, 0};

    for(unsigned int i = 0; i < numDescriptors; i++)
    {
        if(results[i].min < overallDistances.min) overallDistances.min = results[i].min;
        if(results[i].max > overallDistances.max) overallDistances.max = results[i].max;
        overallDistances.avg += (results[i].avg / numOtherDescriptors);


        // std::cout << "distances at " << i << ", min:" << results[i].min << ", max:" << results[i].max << ", avg:" <<  << std::endl;
        // if(results[i].min > (results[i].avg / numOtherDescriptors))
        //     std::cout << "avg distance at " << i << "below min" << std::endl;
        // if(results[i].max < (results[i].avg / numOtherDescriptors))
        //     std::cout << "avg distance at " << i << "above max" << std::endl;         
        // if(results[i].min == 0)
        //     numZero++;
    }

    // std::cout << "Number of zero distances: " << numZero << std::endl;

    std::cout << "Min distance: " << overallDistances.min << std::endl;
    std::cout << "Max distance: " << overallDistances.max << std::endl;
    std::cout << "Avg distance: " << (overallDistances.avg / numDescriptors) << std::endl;

    float min = 0.0f, max = 1024.0f;

    std::vector<float> mins, avgs, maxs;
    mins.reserve(numDescriptors);
    // avgs.reserve(numDescriptors);
    // maxs.reserve(numDescriptors);

    for(size_t i = 0; i < numDescriptors; i++)
    {
        mins.emplace_back(results[i].min);
        // avgs.emplace_back(results[i].avg);
        // maxs.emplace_back(results[i].max);
    }

    showHistogram(mins.data(), numDescriptors, 16, max, min, "../images/json/combined_quicci/histogram_I_min_RICI.json");
    // showHistogram(avgs.data(), numDescriptors, 16, max, min, "../images/json/occlusion/histogram_I_avg.json");
    // showHistogram(maxs.data(), numDescriptors, 16, max, min, "../images/json/occlusion/histogram_I_max.json");

    hipFree(results);
}

__global__
void ComputeCrossDistanceWithThreshold(
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> descriptors, 
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> otherDescriptors,
    ShapeDescriptor::gpu::array<float> thresholds,
    unsigned int *results
)
{
 
    const size_t blockYZIndex = blockIdx.y + blockIdx.z * gridDim.z;
    
    const size_t descriptorIndex = blockIdx.x;
    const size_t otherDescriptorIndex = blockYZIndex < otherDescriptors.length ? blockYZIndex : otherDescriptors.length - 1;
    const int laneIndex = threadIdx.x;


    float threadWeightedHammingDistance = computeQUICCIThreadDistance(descriptors, otherDescriptors, descriptorIndex, otherDescriptorIndex);

    __syncthreads();

    float weightedHammingDistance = warpAllReduceSum(threadWeightedHammingDistance);

    if(laneIndex == 0 && weightedHammingDistance < thresholds[descriptorIndex] && blockYZIndex < otherDescriptors.length)
    {
        atomicAdd(&results[descriptorIndex], 1);
    }
}

void DescriptorDistance::QUICCI::ComputeCrossWiseWithThreshold(
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> needleDescriptors,
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> haystackDescriptors,
    ShapeDescriptor::gpu::array<float> thresholds
)
{
    const unsigned int numDescriptors = needleDescriptors.length;
    const unsigned int numOtherDescriptors = haystackDescriptors.length;

    unsigned int *results;
    hipMallocManaged(&results, numDescriptors * sizeof(unsigned int));

    fillZeros(results, numDescriptors);

    const auto [numY, numZ] = findCorrectDimensions(numOtherDescriptors);

    ComputeCrossDistanceWithThreshold<<<dim3(numDescriptors, numY, numZ), 32 >>>(needleDescriptors, haystackDescriptors, thresholds, results);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    for(unsigned int i = 0; i < numDescriptors; i++)
    {
        std::cout << "Number of distances below threshold " << i << ": " << results[i] << std::endl;
    }

    
    hipFree(results);
}
   

