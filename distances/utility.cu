#include "hip/hip_runtime.h"
#include "utility.cuh"

void showHistogram(float *results, size_t count, size_t rangeCount, float max, float min, std::string outputDir)
{
    std::vector<int> columns(rangeCount, 0);

    float columnRange = (max - min) / rangeCount; 

    for(unsigned int i = 0; i < count; i++)
    {
        size_t column = int((results[i] - min) / columnRange);
        column = column < rangeCount ? column : rangeCount - 1;
        columns.at(column) += 1;
    }

    std::cout << "column count: " << std::endl;

    for(unsigned int i = 0; i < rangeCount; i++)
        std::cout << "column range (" << columnRange * i << ", " << columnRange * (i+1) << "): "  << columns[i] << ", " << std::endl;


    if(!outputDir.empty())
    {
        json jsonfile;

        jsonfile["binValues"] = columns;

        std::ofstream file(outputDir);
        file << jsonfile;
    }
}