#include "hip/hip_runtime.h"
#include "descriptorDistance.cuh"

//taken from https://stackoverflow.com/questions/17399119/how-do-i-use-atomicmax-on-floating-point-values-in-cuda
__device__ __forceinline__ float atomicMinFloat (float * addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMin((int *)addr, __float_as_int(value))) :
            __uint_as_float(atomicMax((unsigned int *)addr, __float_as_uint(value)));

    return old;
}

//taken from https://stackoverflow.com/questions/17399119/how-do-i-use-atomicmax-on-floating-point-values-in-cuda
__device__ __forceinline__ float atomicMaxFloat (float * addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMax((int *)addr, __float_as_int(value))) :
         __uint_as_float(atomicMin((unsigned int *)addr, __float_as_uint(value)));

    return old;
}

__inline__ __device__ unsigned int warpAllReduceSum(unsigned int val) {
    for (int mask = warpSize/2; mask > 0; mask /= 2)
        val += __shfl_xor_sync(0xFFFFFFFF, val, mask);
    return val;
}

__inline__ __device__ float warpAllReduceSum(float val) {
    for (int mask = warpSize/2; mask > 0; mask /= 2)
        val += __shfl_xor_sync(0xFFFFFFFF, val, mask);
    return val;
}

__inline__ __device__ unsigned int getChunkAt(const ShapeDescriptor::QUICCIDescriptor* image, const size_t imageIndex, const int chunkIndex) {
    return image[imageIndex].contents[chunkIndex];
}

__inline__ __device__ int computeImageSumGPU(
        const ShapeDescriptor::QUICCIDescriptor* needleImages,
        const size_t imageIndex) {

    const int laneIndex = threadIdx.x % 32;

    unsigned int threadSum = 0;

    static_assert(spinImageWidthPixels % 32 == 0, "This kernel assumes images are multiples of warp size wide");

    for (int chunk = laneIndex; chunk < uintsPerQUICCImage; chunk += warpSize) {
        unsigned int needleChunk = getChunkAt(needleImages, imageIndex, chunk);
        threadSum += __popc(needleChunk);
    }

    int sum = warpAllReduceSum(threadSum);

    return sum;
}


inline __device__ unsigned int CountBitsTrueLeftFalseRight1(const unsigned int left, const unsigned int right)
{
    return __popc(left & !right);
}

inline __device__ unsigned int CountBitsFalseLeftTrueRight1(const unsigned int left, const unsigned int right)
{
    return __popc(!left & right);
}

inline __device__ unsigned int CountBitsTrueInChunk(const unsigned int chunk)
{
    return __popc(chunk);
}

inline unsigned int GetQUICCIChunk1(const ShapeDescriptor::QUICCIDescriptor* image, const size_t imageIndex, const int chunkIndex)
{
    return image[imageIndex].contents[chunkIndex];
}

void DescriptorDistance::Hamming::CudaComputeDistancesWrapper(
    ShapeDescriptor::QUICCIDescriptor needle, 
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> descriptors, 
    const unsigned int numDescriptors
    )
{

    float * result;
    
    auto start = std::chrono::high_resolution_clock::now();

    hipMallocManaged(&result, numDescriptors * sizeof(float));

    auto now = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(now - start);
    std::cout << "Cuda Malloc duration: " << duration.count() << std::endl;


    start = std::chrono::high_resolution_clock::now();

    CudaComputeDistances<<<numDescriptors, 32>>>(needle, descriptors, numDescriptors, result);


    now = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(now - start);
    std::cout << "Cuda Run Computation: " << duration.count() << std::endl;

    start = std::chrono::high_resolution_clock::now();
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    now = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(now - start);
    std::cout << "Cuda Sync duration: " << duration.count() << std::endl;


    std::cout << "Result: " << (*result / float(numDescriptors)) << std::endl;

    hipFree(result);
    
}

__global__
void DescriptorDistance::Hamming::CudaComputeDistances(
    ShapeDescriptor::QUICCIDescriptor needle, 
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> descriptors, 
    const unsigned int numDescriptors,
    float *result
    )
{
    
    const size_t descriptorIndex = blockIdx.x;
    const int laneIndex = threadIdx.x;

    unsigned int referenceImageBitCount = CountBitsTrueInChunk(needle.contents[laneIndex]);

    unsigned int totalBitsInBitString = spinImageWidthPixels * spinImageWidthPixels;
    unsigned int queryImageUnsetBitCount = totalBitsInBitString - referenceImageBitCount;

    // If any count is 0, bump it up to 1

    #ifdef __HIPCC__
        referenceImageBitCount = max(referenceImageBitCount, 1);
        queryImageUnsetBitCount = max(queryImageUnsetBitCount, 1);
    #else
        referenceImageBitCount = std::max<unsigned int>(referenceImageBitCount, 1);
        queryImageUnsetBitCount = std::max<unsigned int>(queryImageUnsetBitCount, 1);
    #endif

    // The fewer bits exist of a specific pixel type, the greater the penalty for not containing it
    float missedSetBitPenalty = float(totalBitsInBitString) / float(referenceImageBitCount);
    float missedUnsetBitPenalty = float(totalBitsInBitString) / float(queryImageUnsetBitCount);

    unsigned int needleChunk = needle.contents[laneIndex];
    unsigned int haystackChunk = descriptors[descriptorIndex].contents[laneIndex];

    const unsigned int numberOfBitsTrueInNeedleFalseInHaystack = CountBitsTrueLeftFalseRight1(needleChunk, haystackChunk);
    const unsigned int numberOfBitsFalseInNeedleTrueInHaystack = CountBitsFalseLeftTrueRight1(needleChunk, haystackChunk);

    const float distance = float(numberOfBitsTrueInNeedleFalseInHaystack) * missedSetBitPenalty
                            + float(numberOfBitsFalseInNeedleTrueInHaystack) * missedUnsetBitPenalty;

    atomicAdd(result, distance);
    // results[descriptorIndex] = distance;   
}


void DescriptorDistance::Hamming::NumberOfDistancesToRandomDesciptorsLowerThanTrueDistance(
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> descriptors,
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> correspondingDescriptors,
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> randomDescriptors
    )
{

    const unsigned int numDescriptors = (descriptors.length < correspondingDescriptors.length) ? descriptors.length : correspondingDescriptors.length;
    const unsigned int numRandomDescriptors = randomDescriptors.length;

    float *trueDescriptorDistance;
    hipMallocManaged(&trueDescriptorDistance, numDescriptors * sizeof(float));

    ComputeElementWiseCuda<<<numDescriptors, 32>>>(descriptors, correspondingDescriptors, numDescriptors, trueDescriptorDistance);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    unsigned int *results;
    hipMallocManaged(&results, numDescriptors * sizeof(unsigned int));

    for(unsigned int i = 0; i < numDescriptors; i++)
        results[i] = 0;

    const unsigned int numZ = numRandomDescriptors / 65535;
    const unsigned int numY = numRandomDescriptors % 65535;

    CudaCompareDescriptors<<<dim3(numDescriptors, numY, numZ), 32 >>>(descriptors, randomDescriptors, trueDescriptorDistance, results);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    hipFree(trueDescriptorDistance);

    // std::cout << "num wrond: " << numDescriptors << std::endl;
    unsigned int numWrong = 0;
    float avgWrong = 0.0f;
    for(unsigned int i = 0; i < numDescriptors; i++)
    {

        if(results[i] != 0)
        {
            std::cout << "desc i: " << i << ", distance to beat: " << trueDescriptorDistance[i] << ", val: " << results[i] << std::endl;
            numWrong++;
            avgWrong += float(results[i]) / numDescriptors;
        }
    }

    std::cout << "num wrong: " << numWrong << std::endl;
    std::cout << "avg wrong: " << avgWrong << std::endl;

    hipFree(results);

}


void DescriptorDistance::Hamming::FindMinDistance(
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> descriptors, 
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> otherDescriptors
)
{
    
    const unsigned int numDescriptors = descriptors.length;
    const unsigned int numOtherDescriptors = otherDescriptors.length;

    float *results;
    hipMallocManaged(&results, numDescriptors * sizeof(float));

    for(unsigned int i = 0; i < numDescriptors; i++)
        results[i] = 1024.0f;

    const unsigned int numZ = numOtherDescriptors / 65535;
    const unsigned int numY = numOtherDescriptors % 65535;

    FindMinDistanceCuda<<<dim3(numDescriptors, numY, numZ), 32 >>>(descriptors, otherDescriptors, results);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    for(unsigned int i = 0; i < numDescriptors; i++)
        std::cout << results[i] << std::endl;
    
    hipFree(results);
}

__device__ __forceinline__ float computeQUICCIThreadDistance(
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> descriptors, 
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> otherDescriptors,
    const size_t descriptorIndex,
    const size_t otherDescriptorsIndex
)
{
    const int laneIndex = threadIdx.x;

    int referenceImageBitCount = computeImageSumGPU(descriptors.content, descriptorIndex);
    ShapeDescriptor::utilities::HammingWeights hammingWeights = ShapeDescriptor::utilities::computeWeightedHammingWeights(referenceImageBitCount, spinImageWidthPixels * spinImageWidthPixels);

    bool needleImageIsConstant = referenceImageBitCount == 0;

    float threadWeightedHammingDistance = 0;

    auto chunk = laneIndex;

    if(!needleImageIsConstant) 
    {

        unsigned int needleChunk = getChunkAt(descriptors.content, descriptorIndex, chunk);
        unsigned int haystackChunk = getChunkAt(otherDescriptors.content, otherDescriptorsIndex, chunk);

        threadWeightedHammingDistance += ShapeDescriptor::utilities::computeChunkWeightedHammingDistance(hammingWeights, needleChunk, haystackChunk);

    } else 
    {
        unsigned int haystackChunk = getChunkAt(otherDescriptors.content, descriptorIndex, chunk);

        threadWeightedHammingDistance += float(__popc(haystackChunk));
        
    }

    return threadWeightedHammingDistance;
}

__global__
void DescriptorDistance::Hamming::FindMinDistanceCuda(
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> descriptors, 
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> otherDescriptors,
    float *results
)
{
    
    const size_t blockXZIndex = blockIdx.y + blockIdx.z * gridDim.z;
    
    const size_t descriptorIndex = blockIdx.x;
    const size_t randomDescriptorsIndex = blockXZIndex < otherDescriptors.length ? blockXZIndex : otherDescriptors.length - 1;
    const int laneIndex = threadIdx.x;

    float threadWeightedHammingDistance = computeQUICCIThreadDistance(descriptors, otherDescriptors, descriptorIndex, randomDescriptorsIndex);

    __syncthreads();

    float weightedHammingDistance = warpAllReduceSum(threadWeightedHammingDistance);

    if(laneIndex == 0)
        atomicMinFloat(&results[descriptorIndex], weightedHammingDistance);

}


void DescriptorDistance::Hamming::FindDistances(
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> descriptors, 
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> otherDescriptors
)
{
    
    const unsigned int numDescriptors = descriptors.length;
    const unsigned int numOtherDescriptors = otherDescriptors.length;

    distances *results;
    hipMallocManaged(&results, numDescriptors * sizeof(distances));

    for(unsigned int i = 0; i < numDescriptors; i++)
    {
        results[i] = {1024.0f, 0.0f, 0.0f};
    }

    unsigned int numZ = 1;
    unsigned int numY = numOtherDescriptors;

    if(numOtherDescriptors > 65535)
    {
        float numSquareRoot = sqrt(numOtherDescriptors);
        int flooredRoot = int(numSquareRoot);


        numY = flooredRoot;
        numZ = flooredRoot + 1;

        numY += numY * numZ < numOtherDescriptors ? 1 : 0; 
    }
    

    FindDistancesCuda<<<dim3(numDescriptors, numY, numZ), 32 >>>(descriptors, otherDescriptors, results);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    unsigned int numZero = 0;

    for(unsigned int i = 0; i < numDescriptors; i++)
    {
        // std::cout << "distances at " << i << ", min:" << results[i].min << ", max:" << results[i].max << ", avg:" << (results[i].avg / numOtherDescriptors) << std::endl;
        // if(results[i].min > (results[i].avg / numOtherDescriptors))
        //     std::cout << "avg distance at " << i << "below min" << std::endl;
        // if(results[i].max < (results[i].avg / numOtherDescriptors))
        //     std::cout << "avg distance at " << i << "above max" << std::endl;         
        if(results[i].min == 0)
            numZero++;
    }

    std::cout << "Number of zero distances: " << numZero << std::endl;
    
    hipFree(results);
}

__global__
void DescriptorDistance::Hamming::FindDistancesCuda(
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> descriptors, 
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> otherDescriptors,
    distances *results
)
{
 
    const size_t blockXZIndex = blockIdx.y + blockIdx.z * gridDim.z;
    
    const size_t descriptorIndex = blockIdx.x;
    const size_t otherDescriptorIndex = blockXZIndex < otherDescriptors.length ? blockXZIndex : otherDescriptors.length - 1;
    const int laneIndex = threadIdx.x;


    float threadWeightedHammingDistance = computeQUICCIThreadDistance(descriptors, otherDescriptors, descriptorIndex, otherDescriptorIndex);

    __syncthreads();

    float weightedHammingDistance = warpAllReduceSum(threadWeightedHammingDistance);

    if(laneIndex == 0 && blockXZIndex < otherDescriptors.length)
    {
        atomicMinFloat(&results[descriptorIndex].min, weightedHammingDistance);
        atomicMaxFloat(&results[descriptorIndex].max, weightedHammingDistance);
        atomicAdd(&results[descriptorIndex].avg, weightedHammingDistance);
    }
}


void DescriptorDistance::Hamming::FindElementWiseDistances(
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> descriptors, 
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> otherDescriptors, 
    ShapeDescriptor::gpu::array<IndexPair> pairs
)
{

    const unsigned int numPairs = pairs.length;

    float *results;
    hipMallocManaged(&results, numPairs * sizeof(float));


    FindElementWiseDistancesCuda<<<numPairs, 32 >>>(descriptors, otherDescriptors, pairs, results);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    unsigned int numZero = 0;

    for(unsigned int i = 0; i < numPairs; i++)
    {
        std::cout << "distances at " << i << ": " << results[i] << std::endl;   
        if(results[i] == 0)
            numZero++;
    }
    
    std::cout << numZero << std::endl;

    hipFree(results);

}


__global__
void DescriptorDistance::Hamming::FindElementWiseDistancesCuda(
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> descriptors, 
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> otherDescriptors, 
    ShapeDescriptor::gpu::array<IndexPair> pairs,
    float *results
)
{
    const size_t descriptorIndex = pairs[blockIdx.x].left;
    const size_t otherDescriptorIndex = pairs[blockIdx.x].right;
    const int laneIndex = threadIdx.x;

    // printf("pair: %d, %d", static_cast<int>(descriptorIndex), static_cast<int>(otherDescriptorIndex));

    float threadWeightedHammingDistance = computeQUICCIThreadDistance(descriptors, otherDescriptors, descriptorIndex, otherDescriptorIndex);

    __syncthreads();

    float weightedHammingDistance = warpAllReduceSum(threadWeightedHammingDistance);

    if(laneIndex == 0)
        results[descriptorIndex] = weightedHammingDistance;
    

}

__global__ void DescriptorDistance::Hamming::CudaCompareDescriptors(
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> descriptors,
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> randomDescriptors,
    float *trueDescriptorDistance,
    unsigned int *results
    )
{

    const size_t descriptorIndex = blockIdx.x;
    const size_t randomDescriptorsIndex = blockIdx.y + blockIdx.z * 65535;
    const int laneIndex = threadIdx.x;


    int referenceImageBitCount = computeImageSumGPU(descriptors.content, descriptorIndex);
    ShapeDescriptor::utilities::HammingWeights hammingWeights = ShapeDescriptor::utilities::computeWeightedHammingWeights(referenceImageBitCount, spinImageWidthPixels * spinImageWidthPixels);

    bool needleImageIsConstant = referenceImageBitCount == 0;

    float threadWeightedHammingDistance = 0;

    auto chunk = laneIndex;

    if(!needleImageIsConstant) 
    {

        unsigned int needleChunk = getChunkAt(descriptors.content, descriptorIndex, chunk);
        unsigned int haystackChunk = getChunkAt(randomDescriptors.content, randomDescriptorsIndex, chunk);

        threadWeightedHammingDistance += ShapeDescriptor::utilities::computeChunkWeightedHammingDistance(hammingWeights, needleChunk, haystackChunk);

    } else 
    {
        unsigned int haystackChunk = getChunkAt(randomDescriptors.content, descriptorIndex, chunk);

        threadWeightedHammingDistance += float(__popc(haystackChunk));
        
    }
    __syncthreads();

    float weightedHammingDistance = warpAllReduceSum(threadWeightedHammingDistance);

    if(laneIndex == 0 && weightedHammingDistance < trueDescriptorDistance[descriptorIndex])
        atomicAdd(&results[descriptorIndex], 1);

}

void DescriptorDistance::Hamming::ComputeElementWiseCUDAWrapper(
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> descriptors, 
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> otherDescriptors    
    )
{

    float * results;
    
    auto start = std::chrono::high_resolution_clock::now();

    const unsigned int numDescriptors = descriptors.length;
    hipMallocManaged(&results, numDescriptors * sizeof(float));

    auto now = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(now - start);
    std::cout << "Cuda Malloc duration: " << duration.count() << std::endl;


    start = std::chrono::high_resolution_clock::now();

    ComputeElementWiseCuda<<<numDescriptors, 32>>>(descriptors, otherDescriptors, numDescriptors, results);


    now = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(now - start);
    std::cout << "Cuda Run Computation: " << duration.count() << std::endl;

    start = std::chrono::high_resolution_clock::now();
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    now = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(now - start);
    std::cout << "Cuda Sync duration: " << duration.count() << std::endl;

    // unsigned int indexOfMax = 0;
    // float currentMax = 0.0f;

    for(unsigned int i = 0; i < numDescriptors; i++)
    {
        std::cout << "distance at " << i << ": " << results[i] << std::endl;
        // if(results[i] > currentMax) indexOfMax = i;
    }

    hipFree(results);
    
    // std::cout << indexOfMax << std::endl;
}

__global__
void DescriptorDistance::Hamming::ComputeElementWiseCuda(
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> descriptors,
    ShapeDescriptor::gpu::array<ShapeDescriptor::QUICCIDescriptor> otherDescriptors,
    const unsigned int numDescriptors,
    float *results
    )
{
    
    const size_t descriptorIndex = blockIdx.x;
    const int laneIndex = threadIdx.x;

    int referenceImageBitCount = computeImageSumGPU(descriptors.content, descriptorIndex);
    ShapeDescriptor::utilities::HammingWeights hammingWeights = ShapeDescriptor::utilities::computeWeightedHammingWeights(referenceImageBitCount, spinImageWidthPixels * spinImageWidthPixels);


    bool needleImageIsConstant = referenceImageBitCount == 0;

    float threadWeightedHammingDistance = 0;

    auto chunk = laneIndex;

    if(!needleImageIsConstant) 
    {
        unsigned int needleChunk = getChunkAt(descriptors.content, descriptorIndex, chunk);
        unsigned int haystackChunk = getChunkAt(otherDescriptors.content, descriptorIndex, chunk);

        threadWeightedHammingDistance += ShapeDescriptor::utilities::computeChunkWeightedHammingDistance(hammingWeights, needleChunk, haystackChunk);
    } else 
    {
        unsigned int haystackChunk = getChunkAt(otherDescriptors.content, descriptorIndex, chunk);

        threadWeightedHammingDistance += float(__popc(haystackChunk));
        
    }


    float weightedHammingDistance = warpAllReduceSum(threadWeightedHammingDistance);

    if(laneIndex == 0)
        results[descriptorIndex] = weightedHammingDistance;
    
}

void DescriptorDistance::Hamming::testWrapper()
{
    int *n;
    hipMallocManaged(&n, sizeof(int));


    // for(unsigned int i = 0; i < 32; i++)
    //     n[i] = 0;

    test<<<dim3(32,32),32>>>(n);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    // for(unsigned int i = 0; i < 32; i++)
        std::cout << "n: " << n[0] << std::endl;

    hipFree(n);
}

__global__ void DescriptorDistance::Hamming::test(int *n)
{
    const int laneIndex = threadIdx.x;
    const int xIdx = blockIdx.x;
    const int yIdx = blockIdx.y;

    __shared__ int blockSum;

    if(laneIndex == 0)
        blockSum = 0;
    __syncthreads();

    atomicAdd(&blockSum, xIdx);
    __syncthreads();


    if(laneIndex == 0)
        printf("block (%d , %d): %d\n", xIdx, yIdx, (blockSum / 32));
    // n[laneIndex] = laneIndex;
    
}